
#include<iostream>
#include "llvm/ADT/APSInt.h"
#include "llvm/Analysis/ConstantFolding.h"
#include "llvm/IR/CallSite.h"
#include "llvm/IR/Constants.h"
#include "llvm/IR/DebugInfo.h"
#include "llvm/IR/LLVMContext.h"
#include "llvm/IR/Module.h"
#include "llvm/IRReader/IRReader.h"
#include "llvm/Support/CommandLine.h"
#include "llvm/Support/ManagedStatic.h"
#include "llvm/Support/PrettyStackTrace.h"
#include "llvm/Support/Signals.h"
#include "llvm/Support/SourceMgr.h"
#include "llvm/Support/raw_ostream.h"
#include "llvm/IR/InstIterator.h"
#include "/home/g1f/Documents/deft/try_cuda/kernel.cuh"

#include <bitset>
#include <memory>
#include <string>



using namespace llvm;
using std::string;
using std::unique_ptr;


static cl::OptionCategory propagationCategory{"constant propagation options"};

static cl::opt<string> inPath{cl::Positional,
                              cl::desc{"<Module to analyze>"},
                              cl::value_desc{"bitcode filename"},
                              cl::init(""),
                              cl::Required,
                              cl::cat{propagationCategory}};

using namespace std;


int
main(int argc, char** argv) {
  // This boilerplate provides convenient stack traces and clean LLVM exit
  // handling. It also initializes the built in support for convenient
  // command line option handling.
  sys::PrintStackTraceOnErrorSignal(argv[0]);
  llvm::PrettyStackTraceProgram X(argc, argv);
  llvm_shutdown_obj shutdown;
  cl::HideUnrelatedOptions(propagationCategory);
  cl::ParseCommandLineOptions(argc, argv);

  // Construct an IR file from the filename passed on the command line.
  SMDiagnostic err;
  LLVMContext context;
  unique_ptr<Module> module = parseIRFile(inPath.getValue(), err, context);

  if (!module.get()) {
    errs() << "Error reading bitcode file: " << inPath << "\n";
    err.print(argv[0], errs());
    return -1;
  }

  auto* mainFunction = module->getFunction("main");
  if (!mainFunction) {
    llvm::report_fatal_error("Unable to find main function.");
  }

  dim3 gridDim(1);
  dim3 blockDim(1);
  int* b;
  b=new int;
  *b = 1;
  // printf("%d\n", *b);


  int* d_b;llvm::Instruction* d_i;

  hipMalloc((void **)&d_b, sizeof(int));
  hipMalloc((void **)&d_i, sizeof(llvm::Instruction));

  for (auto& i : llvm::instructions(*mainFunction)) {
    hipMemcpy(d_i, &i, sizeof(llvm::Instruction), hipMemcpyHostToDevice);

    kernel<<<gridDim, blockDim>>>(d_i, d_b);

    hipMemcpy(b, d_b, sizeof(int), hipMemcpyDeviceToHost);
    // printf("%d\n", *b);
  }
  
  return 0;
}