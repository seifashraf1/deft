#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void matrixMultiplication(int** M, int* Msummary, int n, int alpha, int omega) {
    // Calculate the indices of the current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform matrix multiplication for the current thread's position
    if (row < n && col < n) {
        int result = M[omega][row * n + col];
        
        for (int i = omega - 1; i >= alpha; i--) {
            result *= M[i][row * n + col];
        }
        
        Msummary[row * n + col] = result;
    }
}

void fillMatrices(int*** h_M, int*** d_M, int numMatrices, int n)
{
    srand(time(NULL));
    *h_M = new int*[numMatrices];
    *d_M = new int*[numMatrices];
    
    for (int m = 0; m < numMatrices; m++)
    {
        (*h_M)[m] = new int[n * n];
        hipMalloc(&((*d_M)[m]), n * n * sizeof(int));

        for (int i = 0; i < n; i++)
        {
            for (int j = 0; j < n; j++)
            {
                (*h_M)[m][i * n + j] = rand() % 2;  // Random value 0 or 1
            }
        }

        hipMemcpy((*d_M)[m], (*h_M)[m], n * n * sizeof(int), hipMemcpyHostToDevice);
    }
}

int main() {
    // Assuming M is a 3D array of matrices, where M[numMatrices][n][n]
    int numMatrices = 5; // Number of matrices
    int n = 10; // Size of each matrix
    int omega = numMatrices - 1; // Value of alpha
    int alpha = 0;
    
    // Allocate memory for the input matrices M on the host
    int*** h_M = new int**[numMatrices];
    for (int i = 0; i < numMatrices; i++) {
        h_M[i] = new int*[n];
        for (int j = 0; j < n; j++) {
            h_M[i][j] = new int[n];
        }
    }
    
    // Fill the input matrices with some values
    srand(time(NULL));
    for (int m = 0; m < numMatrices; m++)
    {
        for (int i = 0; i < n; i++)
        {
            for (int j = 0; j < n; j++)
            {
                h_M[m][i][j] = rand() % 2;  // Random value 0 or 1
            }
        }
    }
    
    // Allocate memory for the input and output matrices on the device
    int*** d_M;
    int* d_Msummary;
    hipMalloc(&d_M, numMatrices * sizeof(int**));
    for (int i = 0; i < numMatrices; i++) {
        hipMalloc(&(d_M[i]), n * n * sizeof(int));
    }
    hipMalloc(&d_Msummary, n * n * sizeof(int));
    
    // Copy the input matrices from the host to the device
    for (int i = 0; i < numMatrices; i++) {
        hipMemcpy(d_M[i], h_M[i], n * n * sizeof(int), hipMemcpyHostToDevice);
    }
    
    dim3 blockSize(16, 16);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y);

    // Allocate memory for d_Msummary
    int** d_Msummary;
    hipMalloc((void**)&d_Msummary, n * sizeof(int*));
    for (int i = 0; i < n; i++)
        hipMalloc((void**)&(d_Msummary[i]), n * sizeof(int));

    // Call the matrix multiplication CUDA kernel
    matrixMultiplication<<<gridSize, blockSize>>>(d_M, d_Msummary, n, alpha, omega);

    
    // Copy the result matrix from the device to the host
    int* h_Msummary = new int[n * n];
    hipMemcpy(h_Msummary, d_Msummary, n * n * sizeof(int), hipMemcpyDeviceToHost);
    
    // Free the allocated memory on the device
    for (int i = 0; i < numMatrices; i++) {
        hipFree(d_M[i]);
    }
    hipFree(d_Msummary);
    
    // Use the resulting Msummary matrix
    
    // Free the allocated memory on the host
    for (int i = 0; i < numMatrices; i++) {
        for (int j = 0; j < n; j++) {
            delete[] h_M[i][j];
        }
        delete[] h_M[i];
    }
    delete[] h_M;
    delete[] h_Msummary;

    return 0;
}